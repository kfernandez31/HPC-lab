#include "hip/hip_runtime.h"
/**
 * stencil.cu: a simple 1d stencil on GPU and on CPU
 * 
 * Implement the basic stencil and make sure it works correctly.
 * Then, play with the code
 * - Experiment with block sizes, various RADIUSes and NUM_ELEMENTS.
 * - Measure the memory transfer time, estimate the effective memory bandwidth.
 * - Estimate FLOPS (floating point operations per second)
 * - Switch from float to double: how the performance changes?
*/

scp -r kk429629@students:/home/students/mismap/k/kk429629/HPC/lab3 ~/courses/HPC/lab3


#include <time.h>
#include <stdio.h>
#include <algorithm>
#include <cassert>
#include <cmath>

#define RADIUS        3
#define NUM_ELEMENTS  1000 
#define BLOCK_WIDTH   10

#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))

static void handleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

__global__ void stencil_1d(float *in, float *out) {
  //PUT YOUR CODE HERE
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < NUM_ELEMENTS) {
    out[i] = 0;
    int start = std::max(0, i - RADIUS);
    int end   = std::min(NUM_ELEMENTS - 1, i + RADIUS);
    for (int j = start; j <= end; j++) {
      out[i] += in[j];
    }
  }
}

void cpu_stencil_1d(float *in, float *out) {
  //PUT YOUR CODE HERE
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    int start = std::max(0, i - RADIUS);
    int end   = std::min(NUM_ELEMENTS - 1, i + RADIUS);
    for (int j = start; j <= end; j++) {
      out[i] += in[j];
    }
  }
}

bool check(float *out, float *dev_out) {
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    if (std::abs(out[i] - dev_out[i]) > std::numeric_limits<float>::epsilon()) {
      printf("Mismatch on index %d (expected = %f, actual = %f)\n", i, out[i], dev_out[i]);
      return false;
    }
  }
  return true;
}

int main() {
  //PUT YOUR CODE HERE - INPUT AND OUTPUT ARRAYS
  float in[NUM_ELEMENTS], out[NUM_ELEMENTS], dev_out_cpy[NUM_ELEMENTS];
  float *dev_in, *dev_out;

  for (int i = 0; i < NUM_ELEMENTS; i++) {
    in[i] = 1;
  }
  
  hipEvent_t gpu_start, gpu_stop;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);
  hipEventRecord(gpu_start, 0);

  //PUT YOUR CODE HERE - DEVICE MEMORY ALLOCATION
  hipMalloc((void**)&dev_in,  NUM_ELEMENTS * sizeof(float));
  hipMalloc((void**)&dev_out, NUM_ELEMENTS * sizeof(float));

  hipMemcpy(dev_in,  in,  NUM_ELEMENTS * sizeof(float), hipMemcpyHostToDevice);

  //PUT YOUR CODE HERE - KERNEL EXECUTION
  int num_blocks = (NUM_ELEMENTS + BLOCK_WIDTH - 1) / BLOCK_WIDTH;
  stencil_1d<<<num_blocks, NUM_ELEMENTS>>>(dev_in, dev_out);

  cudaCheck(hipPeekAtLastError());

  //PUT YOUR CODE HERE - COPY RESULT FROM DEVICE TO HOST
  struct timespec memtransfer_start, memtransfer_stop;  
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &memtransfer_start);
  hipMemcpy(dev_out_cpy, dev_out, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &memtransfer_stop);
  double memoryTransferElapsedTime = (memtransfer_stop.tv_sec - memtransfer_start.tv_sec) * 1e3 + (memtransfer_stop.tv_nsec - memtransfer_start.tv_nsec) / 1e6;
  printf("Memory transfer time:  %.4f ms\n", memoryTransferElapsedTime);
  printf("Bandwidth:  %.4f Mb/s\n", 1e3 * NUM_ELEMENTS * sizeof(float) / memoryTransferElapsedTime);
  printf("FLOPS: %.4f\n", 1e3 * NUM_ELEMENTS * (2 * RADIUS + 1) / memoryTransferElapsedTime);

  hipEventRecord(gpu_stop, 0);
  hipEventSynchronize(gpu_stop);
  float gpuElapsedTime;
  hipEventElapsedTime(&gpuElapsedTime, gpu_start, gpu_stop);
  printf("Total GPU execution time:  %.4f ms\n", gpuElapsedTime);
  printf("Memory transfer time:  %.4f ms\n", gpuElapsedTime);
  hipEventDestroy(gpu_start);
  hipEventDestroy(gpu_stop);

  //PUT YOUR CODE HERE - FREE DEVICE MEMORY  
  hipFree(dev_in);
  hipFree(dev_out);

  struct timespec cpu_start, cpu_stop;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);
 
  cpu_stencil_1d(in, out);

  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
  double cpuElapsedTime = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
  printf("CPU execution time:  %.4f ms\n", cpuElapsedTime);

  

  if (!check(out, dev_out_cpy)) {
    return 1;
  } else {
    printf("OK\n");
    return 0;
  }
}
